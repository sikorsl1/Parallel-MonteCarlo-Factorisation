#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

__device__ unsigned long long int gcd(unsigned long long int a, unsigned long long int b){
	unsigned long long int r=0;
	while(b!=0){
		r = a%b;
		a = b;
		b = r;
	}
	return a;
}

__global__ void MonteCarlo(unsigned long long int n, unsigned long long int *d){
	unsigned long long int dtmp = 1;
	unsigned long long int a = threadIdx.x;
	unsigned long long int b = threadIdx.x;
	while((dtmp==1||dtmp==n) && (*d==1||*d==n)){
		a = a*a+a+1;
		b = b*b+b+1;
		b = b*b+b+1;
		dtmp = gcd(a-b,n);
	}
	*d=dtmp;
}

int main(int argc, char *argv[]){
	if(argc<2)
		exit(0);
	unsigned long long int n = atoll(argv[1]);
	unsigned long long int *ptrd;
	unsigned long long int d = 1;
	hipMalloc((void**) &ptrd, sizeof(unsigned long long int));
	hipMemcpy(ptrd, &d, sizeof(unsigned long long int),hipMemcpyHostToDevice);

	MonteCarlo<<<1,5>>>(n,ptrd);

	hipMemcpy(&d, ptrd, sizeof(unsigned long long int),hipMemcpyDeviceToHost);

	printf("%lld\n",d);
	hipFree(ptrd);
	return 0;
}
